#include "hip/hip_runtime.h"
/*
 * kmeans_gpu.cu
 */

#include <hip/hip_runtime_api.h>
#include <iostream>
#include <iomanip>
#include <stdio.h>

#include "util.h"

using namespace std;

// declarations

static int ThreadsPerBlock = 512;

static inline int nextPowerOfTwo(int n);

__device__ inline static float euclid_dist_2(const unsigned int size, const long count, const unsigned int k,
		float *points, float *clusters, int oId, int cId);

__global__ void compute_delta(int *deviceIntermediates, int numIntermediates, int numIntermediates2);

__global__ void nearest_cluster(const unsigned int size, const long count, const unsigned int k, float *points,
		float *deviceClusters, int *membership, int *intermediates);

// function definitions

static inline int nextPowerOfTwo(int n) {
	n--;

	n = n >> 1 | n;
	n = n >> 2 | n;
	n = n >> 4 | n;
	n = n >> 8 | n;
	n = n >> 16 | n;
//  n = n >> 32 | n

	return ++n;
}

__device__ inline static
float euclid_dist_2(const unsigned int size, const long count, const unsigned int k, float *points, float *clusters,
		int oId, int cId) {
	float ans = 0.0;

	for (int i = 0; i < size; i++) {
		ans += (points[count * i + oId] - clusters[k * i + cId]) * (points[count * i + oId] - clusters[k * i + cId]);
	}

	return ans;
}

__global__
void compute_delta(int *deviceIntermediates, int numIntermediates, int numIntermediates2) {
	extern __shared__ unsigned int intermediates[];
	intermediates[threadIdx.x] = (threadIdx.x < numIntermediates) ? deviceIntermediates[threadIdx.x] : 0;

	__syncthreads();

	for (unsigned int s = numIntermediates2 / 2; s > 0; s >>= 1) {
		if (threadIdx.x < s) {
			intermediates[threadIdx.x] += intermediates[threadIdx.x + s];
		}
		__syncthreads();
	}

	if (threadIdx.x == 0) {
		deviceIntermediates[0] = intermediates[0];
	}
}

__global__ void nearest_cluster(const unsigned int size, const long count, const unsigned int k, float *points,
		float *deviceClusters, int *membership, int *intermediates) {
	extern __shared__ char sharedMemory[];
	unsigned char *membershipChanged = (unsigned char *) sharedMemory;
	int objectId = blockDim.x * blockIdx.x + threadIdx.x;
	float *clusters = deviceClusters;

	membershipChanged[threadIdx.x] = 0;

	if (objectId < count) {
		int index = 0;
		float dist;
		float min_dist = euclid_dist_2(size, count, k, points, clusters, objectId, 0);

		for (int i = 1; i < k; i++) {
			dist = euclid_dist_2(size, count, k, points, clusters, objectId, i);

			if (dist < min_dist) {
				min_dist = dist;
				index = i;
			}
		}

		if (membership[objectId] != index) {
			membershipChanged[threadIdx.x] = 1;
		}

		membership[objectId] = index;

		__syncthreads();    //  For membershipChanged[]

		for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
			if (threadIdx.x < s) {
				membershipChanged[threadIdx.x] += membershipChanged[threadIdx.x + s];
			}
			__syncthreads();
		}

		if (threadIdx.x == 0) {
			intermediates[blockIdx.x] = membershipChanged[0];
		}
	}
}

void kmeans_global_gpu(float **points, float ** clusters, int *membership, const long count, const unsigned int size,
		const unsigned int k) {
	int index, loop = 0;
	int *newClusterSize = (int*) calloc(k, sizeof(int));
	float delta;
	float threshold = 0.0002;
	float **dimObjects = malloc2D(size, count);
	float **dimClusters = malloc2D(size, k);
	float **newClusters = malloc2D(size, k);

	float *deviceObjects;
	float *deviceClusters;
	int *deviceMembership;
	int *deviceIntermediates;

	double startTime = time();

	hipSetDevice(0);

	for (int i = 0; i < size; i++) {
		for (int j = 0; j < count; j++)
			dimObjects[i][j] = points[j][i];
		for (int j = 0; j < k; j++)
			dimClusters[i][j] = dimObjects[i][j];
	}

	for (int i = 0; i < count; i++) {
		membership[i] = -1;
	}

	memset(newClusters[0], 0, size * k * sizeof(float));

	const unsigned int numBlocks = (count + ThreadsPerBlock - 1) / ThreadsPerBlock;
	const unsigned int clusterBlockSharedDataSize = numBlocks * sizeof(unsigned char);

	cout << "  Blocks: " << numBlocks << endl;
	cout << "  Threads: " << ThreadsPerBlock << endl;

	const unsigned int numReductionThreads = nextPowerOfTwo(numBlocks);
	const unsigned int reductionBlockSharedDataSize = numReductionThreads * sizeof(unsigned int);

	hipMalloc((void **) &deviceClusters, k * size * sizeof(float));
	hipMalloc((void **) &deviceObjects, count * size * sizeof(float));
	hipMalloc((void **) &deviceMembership, count * sizeof(int));
	hipMalloc((void **) &deviceIntermediates, numReductionThreads * sizeof(unsigned int));

	hipMemcpy(deviceObjects, dimObjects[0], count * size * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(deviceMembership, membership, count * sizeof(int), hipMemcpyHostToDevice);

	do {
		hipMemcpy(deviceClusters, dimClusters[0], k * size * sizeof(float), hipMemcpyHostToDevice);

		nearest_cluster<<<numBlocks, ThreadsPerBlock, clusterBlockSharedDataSize>>>(size, count, k, deviceObjects,
				deviceClusters, deviceMembership, deviceIntermediates);

		compute_delta<<<1, numReductionThreads, reductionBlockSharedDataSize>>>(deviceIntermediates, numBlocks,
				numReductionThreads);

		hipDeviceSynchronize();

		hipMemcpy(membership, deviceMembership, count * sizeof(int), hipMemcpyDeviceToHost);

		for (int i = 0; i < count; i++) {
			index = membership[i];

			newClusterSize[index]++;
			for (int j = 0; j < size; j++) {
				newClusters[j][index] += points[i][j];
			}
		}

		for (int i = 0; i < k; i++) {
			for (int j = 0; j < size; j++) {
				if (newClusterSize[i] > 0) {
					dimClusters[j][i] = newClusters[j][i] / newClusterSize[i];
				}
				newClusters[j][i] = 0.0;
			}
			newClusterSize[i] = 0;
		}

		int d;
		hipMemcpy(&d, deviceIntermediates, sizeof(int), hipMemcpyDeviceToHost);
		delta = (float) d / count;
	} while (delta > threshold && loop++ < 500);

	for (int i = 0; i < k; i++) {
		for (int j = 0; j < size; j++) {
			clusters[i][j] = dimClusters[j][i];
		}
	}

	double endTime = time() - startTime;
	cout << "Time: " << endTime << " with " << loop << " iterations (";
	cout << std::setprecision(2) << (delta * 100) << "% changed)" << endl;

	hipFree(deviceObjects);
	hipFree(deviceClusters);
	hipFree(deviceMembership);
	hipFree(deviceIntermediates);

	free(dimObjects[0]);
	free(dimClusters[0]);
	free(newClusters[0]);

	free(dimObjects);
	free(dimClusters);
	free(newClusters);
	free(newClusterSize);
}
