/*
 * util.cpp
 */

#include <dirent.h>
#include <stdlib.h>
#include <fstream>
#include <iomanip>
#include <iostream>
#include <math.h>
#include <sstream>
#include <iterator>
#include <vector>
#include <sys/time.h>
#include <sys/stat.h>
#include <stdio.h>

#include <opencv2/core/core.hpp>

#include "Test.h"

using namespace std;

// functions

void initCUDA() {
	int deviceCount = 0;
	hipGetDeviceCount(&deviceCount);
	if (deviceCount == 0) {
		cout << "Error: no CUDA device found.\n" << endl;
		exit(1);
	}

	hipSetDevice(0);
}

void checkCUDAError(const char* functionName) {
	hipError_t error = hipGetLastError();
	if (error != hipSuccess) {
		cout << "CUDA error in " << functionName << ": " << hipGetErrorString(error) << endl;
		exit(1);
	}
}

float** malloc2D(int rows, int cols) {
	float** array = (float **) malloc(rows * sizeof(float*));
	array[0] = (float *) malloc(rows * cols * sizeof(float));
	for (int i = 1; i < rows; i++)
		array[i] = array[i - 1] + cols;
	return array;
}

float** matToPtr(cv::Mat *features) {
	float **points = (float**) malloc(features->rows * sizeof(float*));

	for (int i = 0; i < features->rows; i++) {
		points[i] = (float*) malloc(sizeof(float) * features->cols);
		float *row = features->ptr<float>(i);

		for (int j = 0; j < features->cols; j++) {
			points[i][j] = row[j];
		}
	}

	return points;
}

template<typename T>
void print_array(T& array, int m, int n) {
	for (int i = 0; i < m; i++) {
		for (int j = 0; j < n; j++) {
			typename T::value_type value = array[i * n + j];
			std::cout << value << " ";
		}
		std::cout << std::endl;
	}
}

void printHistogram(string imagePath, float *histo, int k) {
	cout << "Word frequencies for " << imagePath << ": " << endl;

	for (int i = 0; i < k; i++) {
		cout << "bin " << i << ": " << histo[i] << endl;
	}
}

vector<string> split(const string s, char delim) {
	vector<string> result;
	stringstream ss;
	ss.str(s);
	string item;

	while (getline(ss, item, delim)) {
		result.push_back(item);
	}
	return result;
}

vector<string> readDir(string directory) {
	DIR *dir;
	vector<string> out;

	class dirent *ent;
	class stat st;

	dir = opendir(directory.c_str());
	while ((ent = readdir(dir)) != NULL) {
		const string file_name = ent->d_name;
		const string full_file_name = directory + "/" + file_name;

		if (file_name[0] == '.')
			continue;

		if (stat(full_file_name.c_str(), &st) == -1)
			continue;

		const bool is_directory = (st.st_mode & S_IFDIR) != 0;

		if (is_directory)
			continue;

		out.push_back(full_file_name);
	}
	closedir(dir);
	return out;
}

double time () {
    struct timeval start;
    struct timezone timezone;

    if (gettimeofday(&start, &timezone) == -1) {
        perror("Error: calling gettimeofday() not successful.\n");
    }
    		/* in seconds */			/* in microseconds */
    return ((double) start.tv_sec) + ((double) start.tv_usec) / 1000000.0;
}
