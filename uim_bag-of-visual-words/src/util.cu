/*
 * util.cpp
 */

#include <stdlib.h>
#include <iomanip>
#include <iostream>
#include <math.h>
#include <sstream>
#include <iterator>
#include <vector>
#include <sys/time.h>
#include <stdio.h>

#include <opencv2/core/core.hpp>

using namespace std;

// functions

void initCUDA() {
	int deviceCount = 0;
	hipGetDeviceCount(&deviceCount);
	if (deviceCount == 0) {
		cout << "Error: no CUDA device found.\n" << endl;
		exit(1);
	}

	hipSetDevice(0);
}

void checkCUDAError(const char* functionName) {
	hipError_t error = hipGetLastError();
	if (error != hipSuccess) {
		cout << "CUDA error in " << functionName << ": " << hipGetErrorString(error) << endl;
		exit(1);
	}
}

float** malloc2D(int rows, int cols) {
	float** array = (float **) malloc(rows * sizeof(float*));
	array[0] = (float *) malloc(rows * cols * sizeof(float));
	for (int i = 1; i < rows; i++)
		array[i] = array[i - 1] + cols;
	return array;
}

float** matToPtr(cv::Mat *features) {
	float **points = (float**) malloc(features->rows * sizeof(float*));

	for (int i = 0; i < features->rows; i++) {
		points[i] = (float*) malloc(sizeof(float) * features->cols);
		float *row = features->ptr<float>(i);

		for (int j = 0; j < features->cols; j++) {
			points[i][j] = row[j];
		}
	}

	return points;
}

template<typename T>
void print_array(T& array, int m, int n) {
	for (int i = 0; i < m; i++) {
		for (int j = 0; j < n; j++) {
			typename T::value_type value = array[i * n + j];
			std::cout << value << " ";
		}
		std::cout << std::endl;
	}
}

void printHistogram(int* histogram, int bins) {
	cout << "Histogram" << endl;
	int sum = 0;

	for (int i = 0; i < bins; i++) {
		sum += histogram[i];
		cout << histogram[i] << " ";
	}

	cout << endl << "sum: " << sum << endl;
}

vector<string> split(const string s, char delim) {
	vector<string> result;
	stringstream ss;
	ss.str(s);
	string item;

	while (getline(ss, item, delim)) {
		result.push_back(item);
	}
	return result;
}

double time () {
    struct timeval start;
    struct timezone timezone;

    if (gettimeofday(&start, &timezone) == -1) {
        perror("Error: calling gettimeofday() not successful.\n");
    }
    		/* in seconds */			/* in microseconds */
    return ((double) start.tv_sec) + ((double) start.tv_usec) / 1000000.0;
}
