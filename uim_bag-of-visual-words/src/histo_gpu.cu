#include "hip/hip_runtime.h"
/*
 * histo_kernel.cu
 */

#include <hip/hip_runtime_api.h>
#include <iostream>
#include <stdio.h>

#include "util.h"

using namespace std;

// declarations

// Number of Threads per Block
int NumberOfThreadsPerBlock = 1024;

/*
 * The histogram kernel computes the index of the nearest cluster of <clusters> for each feature of <features> on the
 * GPU. The value of <histogram> at the computed index is incremented. <bins> is the number of bins of the histogram,
 * <count> is the number of features and <size> the dimension of each feature / cluster.
 */
__global__ void histo_kernel(float *features, float *clusters, unsigned int *histo, const unsigned int bins,
		const long count, const unsigned int size);

/*
 * Computes the euclidean distance between <point1> and <point2> on the GPU and returns it as a float. Both points
 * must have <size> many dimensions.
 */
__device__ float euclidean_distance_2_gpu(float *point1, float *point2, const unsigned int size);

/**
 * Computes the index of nearest cluster of <clusters> to <point> on the GPU. Both the clusters and the point must have
 * <size> many dimensions. <clusters> is an array of size <k>. To obtain the nearest cluster, the euclidean distance is
 * measured.
 */
__device__ int nearest_cluster_gpu(float *point, const unsigned int size, float *clusters, const unsigned int k);

// function definitions

void histo_gpu(float **features, float **clusters, float *histo, const unsigned int k, const long count,
		const unsigned int size) {
	int numberOfBlocksInGrid = count / NumberOfThreadsPerBlock;
	dim3 dimGrid(numberOfBlocksInGrid, 1);
	dim3 dimBlock(NumberOfThreadsPerBlock, 1);

	// Use an integer histo to sum the total counts for each bin
	unsigned int *intHisto = (unsigned int*) new unsigned int[k];

	// Arrays on the GPU
	unsigned int *deviceHisto = 0;
	float *deviceFeatures = 0;
	float *deviceClusters = 0;

	// Calculate memory needed for data on the GPU
	size_t featureMem = sizeof(float) * count * size;
	size_t clusterMem = sizeof(float) * k * size;
	size_t histoMem = sizeof(unsigned int) * k;
	size_t sharedMem = sizeof(float) * k;

	// Convert the two dimensional clusters and features array to a one dimensional array.
	float *featureArray = (float *) new float[count * size];
	float *clusterArray = (float *) new float[k * size];

	for (int i = 0; i < count; i++) {
		for (int j = 0; j < size; j++)
			featureArray[i * size + j] = features[i][j];
	}

	for (int i = 0; i < k; i++) {
		for (int j = 0; j < size; j++)
			clusterArray[i * size + j] = clusters[i][j];
	}

	// Assure at least one block in the grid
	if (numberOfBlocksInGrid < 1) {
		numberOfBlocksInGrid = 1;
	}

	// Assume we have a cuda device...
	hipSetDevice(0);

	// Allocate memory on the GPU and copy data. Important: Set deviceHisto to all 0!
	hipMalloc((void**) &deviceFeatures, featureMem);
	hipMalloc((void**) &deviceClusters, clusterMem);
	hipMalloc((void**) &deviceHisto, histoMem);
	hipMemcpy(deviceFeatures, featureArray, featureMem, hipMemcpyHostToDevice);
	hipMemcpy(deviceClusters, clusterArray, clusterMem, hipMemcpyHostToDevice);
	hipMemset(deviceHisto, 0, histoMem);

	// TODO: DEBUG
	//cout << "Starting histogram kernel" << endl;
	//cout << "  Blocks: " << numberOfBlocksInGrid << endl;
	//cout << "  Threads: " << NumberOfThreadsPerBlock << endl;

	// kernel invocation
	histo_kernel<<<numberOfBlocksInGrid, dimBlock, sharedMem>>>(deviceFeatures, deviceClusters, deviceHisto, k, count,
			size);
	checkCUDAError("createHistogram - Kernel");
	hipDeviceSynchronize();

	// Copy results from device to host
	hipMemcpy(intHisto, deviceHisto, histoMem, hipMemcpyDeviceToHost);

	// Compute frequencies as floats from the total counts in intHisto
	for (int i = 0; i < k; i++) {
		histo[i] = ((float) intHisto[i]) / count;
	}

	// Free everything
	hipFree((void*) deviceFeatures);
	hipFree((void*) deviceClusters);
	hipFree((void*) deviceHisto);
	free(featureArray);
	free(clusterArray);
	free(intHisto);

	deviceFeatures = 0;
	deviceClusters = 0;
	deviceHisto = 0;
	featureArray = 0;
	clusterArray = 0;
	intHisto = 0;
}

__global__ void histo_kernel(float *features, float *clusters, unsigned int *histo, unsigned int bins, long count,
		unsigned int size) {
	// Allocate shared memory per block for private histogram values
	extern __shared__ int sharedMemory[];
	unsigned int *histo_private = (unsigned int *) sharedMemory;

	if (threadIdx.x < bins) {
		histo_private[threadIdx.x] = 0;
	}

	__syncthreads();

	// Compute global index and offset (size of block)
	unsigned int i = threadIdx.x + blockIdx.x * blockDim.x;
	unsigned int stride = blockDim.x * gridDim.x;

	// Compute for the nearsest cluster and therefore the nearest bin and increment the
	// corresponding bin in the private histogram
	while (i < count) {
		float *feature = &features[i * size];
		int bin = nearest_cluster_gpu(feature, size, clusters, bins);
		atomicAdd(&(histo_private[bin]), 1);
		i += stride;
	}

	__syncthreads();

	// The first bin threads are the "master" threads and accumulate the private values
	// of the shared histograms into the global
	if (threadIdx.x < bins) {
		atomicAdd(&(histo[threadIdx.x]), histo_private[threadIdx.x]);
	}
}

__device__ float euclidean_distance_2_gpu(float *point1, float *point2, const unsigned int size) {
	float dist = 0.f;

	for (int i = 0; i < size; i++) {
		dist += (point1[i] - point2[i]) * (point1[i] - point2[i]);
	}
	return dist;
}

__device__ int nearest_cluster_gpu(float *point, const unsigned int size, float *clusters, const unsigned int k) {
	int index = 0;
	float minDist = euclidean_distance_2_gpu(point, clusters, size);

	for (int i = 1; i < k; i++) {
		float dist = euclidean_distance_2_gpu(point, &clusters[i * size], size);

		if (dist < minDist) { // square root?
			minDist = dist;
			index = i;
		}
	}
	return index;
}
