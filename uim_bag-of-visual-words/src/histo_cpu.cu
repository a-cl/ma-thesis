/*
 * histo_cpu.cu
 */

#include <iostream>
#include <stdio.h>

#include "kmeans_cpu.h"

using namespace std;

// function definitions

void histo_cpu(float **features, float** clusters, float* histo, const unsigned int k, const long count,
		const unsigned int size) {
	for (int i = 0; i < count; i++) {
		float *descriptor = features[i];
		int index = nearest_cluster_cpu(descriptor, size, clusters, k);
		histo[index] = histo[index] + 1;
	}

	for (int i = 0; i < k; i++) {
		histo[i] = histo[i] / count;
	}
}
